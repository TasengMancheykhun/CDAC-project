#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

void diagonalize_symmetric_matrix()
{
    // Define the symmetric matrix (NxN)
    const int N = 3;
    double h_A[N * N] = { 4.0, 1.0, 1.0,   // Row 1
                          1.0, 3.0, 2.0,   // Row 2
                          1.0, 2.0, 5.0 }; // Row 3 (Symmetric)

    double *d_A, *d_W; // Device memory for matrix and eigenvalues
    hipError_t cudaStat;
    hipsolverStatus_t cusolverStat;
    
    // Allocate memory on the device
    cudaStat = hipMalloc((void**)&d_A, N * N * sizeof(double));
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "CUDA malloc failed!\n");
        return;
    }

    cudaStat = hipMalloc((void**)&d_W, N * sizeof(double));
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "CUDA malloc failed!\n");
        hipFree(d_A);
        return;
    }

    // Copy matrix A to device
    cudaStat = hipMemcpy(d_A, h_A, N * N * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "CUDA memcpy failed!\n");
        hipFree(d_A);
        hipFree(d_W);
        return;
    }

    // Create cuSOLVER handle
    hipsolverHandle_t cusolverH;
    cusolverStat = hipsolverDnCreate(&cusolverH);
    if (cusolverStat != HIPSOLVER_STATUS_SUCCESS) {
        fprintf(stderr, "cuSOLVER handle creation failed!\n");
        hipFree(d_A);
        hipFree(d_W);
        return;
    }

    // Query buffer size
    int Lwork;
    cusolverStat = hipsolverDnDsyevd_bufferSize(cusolverH, HIPSOLVER_EIG_MODE_VECTOR,
                                               HIPBLAS_FILL_MODE_LOWER, N, d_A, N, d_W, &Lwork);
    if (cusolverStat != HIPSOLVER_STATUS_SUCCESS) {
        fprintf(stderr, "Buffer size query failed!\n");
        hipsolverDnDestroy(cusolverH);
        hipFree(d_A);
        hipFree(d_W);
        return;
    }

    // Allocate workspace
    double *d_work;
    cudaStat = hipMalloc((void**)&d_work, Lwork * sizeof(double));
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "CUDA malloc for workspace failed!\n");
        hipsolverDnDestroy(cusolverH);
        hipFree(d_A);
        hipFree(d_W);
        return;
    }

    int *d_info;
    cudaStat = hipMalloc((void**)&d_info, sizeof(int));
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "CUDA malloc for info failed!\n");
        hipFree(d_work);
        hipsolverDnDestroy(cusolverH);
        hipFree(d_A);
        hipFree(d_W);
        return;
    }

    // Compute eigenvalues and eigenvectors
    cusolverStat = hipsolverDnDsyevd(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_LOWER,
                                    N, d_A, N, d_W, d_work, Lwork, d_info);
    if (cusolverStat != HIPSOLVER_STATUS_SUCCESS) {
        fprintf(stderr, "Eigen decomposition failed!\n");
        hipFree(d_info);
        hipFree(d_work);
        hipsolverDnDestroy(cusolverH);
        hipFree(d_A);
        hipFree(d_W);
        return;
    }

    // Copy results back to host
    double h_W[N], h_V[N * N];
    cudaStat = hipMemcpy(h_W, d_W, N * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "CUDA memcpy for eigenvalues failed!\n");
    }

    cudaStat = hipMemcpy(h_V, d_A, N * N * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "CUDA memcpy for eigenvectors failed!\n");
    }

    int info;
    cudaStat = hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStat != hipSuccess) {
        fprintf(stderr, "CUDA memcpy for info failed!\n");
    }

    if (info == 0) {
        printf("Eigenvalues:\n");
        for (int i = 0; i < N; i++)
            printf("%f ", h_W[i]);
        printf("\n\nEigenvectors:\n");
        for (int i = 0; i < N; i++) {
            for (int j = 0; j < N; j++)
                printf("%f ", h_V[i * N + j]);
            printf("\n");
        }
    } else {
        fprintf(stderr, "Eigen decomposition failed with INFO = %d\n", info);
    }

    // Cleanup
    hipFree(d_A);
    hipFree(d_W);
    hipFree(d_work);
    hipFree(d_info);
    hipsolverDnDestroy(cusolverH);
}

int main()
{
    diagonalize_symmetric_matrix();
    return 0;
}
